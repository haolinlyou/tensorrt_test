#include "hip/hip_runtime.h"
/**
**********************************************************************************************************************************************************************************************************************************
* @file:	cuda_mul_test.cu
* @author:	lk
* @email:	lk123400@163.com
* @date:	2021-06-20 18:36:47 Sunday
* @brief:	
**********************************************************************************************************************************************************************************************************************************
**/

#include "hip/hip_runtime.h"
#include "timeutils.hpp"
#include <iostream>

using namespace std;


void get_property() 
{
    int dev = 0;
    hipDeviceProp_t devProp;
    //CHECK(hipGetDeviceProperties(&devProp, dev));
    std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;

}


__global__ void mul(float *x, float *y, float *z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    // printf("%d\t%d\t%d\t%d\t%d\t%d\n", threadIdx.x, threadIdx.y, blockDim.x, gridDim.x, stride);
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] * y[i];
    }
}


void test()
{
    int N = 1 << 20;
    int nBytes = N * sizeof(float);

    // 申请托管内存
    float *x, *y, *z;
    hipMallocManaged((void**)&x, nBytes);
    hipMallocManaged((void**)&y, nBytes);
    hipMallocManaged((void**)&z, nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i)
    {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    cout << "threadIdx.x\tblockIdx.x\tblockDim.x\tgridDim.x\tstride\n";
    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

    printf("%d %d \n", gridSize, blockSize);
    // 执行kernel
    mul << < gridSize, blockSize >> >(x, y, z, N);

    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    std::cout << "最大误差: " << maxError << std::endl;

    // 释放内存
    hipFree(x);
    hipFree(y);
    hipFree(z);
}

int main()
{

    get_property();
    
    TimeUtil t;
    t.startTimer();
    test();
    cout << "cost time: " << t.getDuration<TimeUnit::MILLISEC>() << " ms" << endl;

    return 0;
}